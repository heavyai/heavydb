#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
hipStream_t getQueryEngineCudaStream();

#include "BufferCompaction.h"
#include "GpuInitGroups.h"
#include "GpuRtConstants.h"
#include "Logger/Logger.h"

#define checkCudaErrors(err) CHECK_EQ(err, hipSuccess)

template <typename T>
__device__ int8_t* init_columnar_buffer(T* buffer_ptr,
                                        const T init_val,
                                        const uint32_t entry_count,
                                        const int32_t start,
                                        const int32_t step) {
  for (int32_t i = start; i < entry_count; i += step) {
    buffer_ptr[i] = init_val;
  }
  return reinterpret_cast<int8_t*>(buffer_ptr + entry_count);
}

extern "C" __device__ void init_columnar_group_by_buffer_gpu_impl(
    int64_t* groups_buffer,
    const int64_t* init_vals,
    const uint32_t groups_buffer_entry_count,
    const uint32_t key_count,
    const uint32_t agg_col_count,
    const int8_t* col_sizes,
    const bool need_padding,
    const bool keyless,
    const int8_t key_size) {
  const int32_t start = blockIdx.x * blockDim.x + threadIdx.x;
  const int32_t step = blockDim.x * gridDim.x;

  int8_t* buffer_ptr = reinterpret_cast<int8_t*>(groups_buffer);
  if (!keyless) {
    for (uint32_t i = 0; i < key_count; ++i) {
      switch (key_size) {
        case 1:
          buffer_ptr = init_columnar_buffer<int8_t>(
              buffer_ptr, EMPTY_KEY_8, groups_buffer_entry_count, start, step);
          break;
        case 2:
          buffer_ptr =
              init_columnar_buffer<int16_t>(reinterpret_cast<int16_t*>(buffer_ptr),
                                            EMPTY_KEY_16,
                                            groups_buffer_entry_count,
                                            start,
                                            step);
          break;
        case 4:
          buffer_ptr =
              init_columnar_buffer<int32_t>(reinterpret_cast<int32_t*>(buffer_ptr),
                                            EMPTY_KEY_32,
                                            groups_buffer_entry_count,
                                            start,
                                            step);
          break;
        case 8:
          buffer_ptr =
              init_columnar_buffer<int64_t>(reinterpret_cast<int64_t*>(buffer_ptr),
                                            EMPTY_KEY_64,
                                            groups_buffer_entry_count,
                                            start,
                                            step);
          break;
        default:
          // FIXME(miyu): CUDA linker doesn't accept assertion on GPU yet right now.
          break;
      }
      buffer_ptr = align_to_int64(buffer_ptr);
    }
  }
  int32_t init_idx = 0;
  for (int32_t i = 0; i < agg_col_count; ++i) {
    if (need_padding) {
      buffer_ptr = align_to_int64(buffer_ptr);
    }
    switch (col_sizes[i]) {
      case 1:
        buffer_ptr = init_columnar_buffer<int8_t>(
            buffer_ptr, init_vals[init_idx++], groups_buffer_entry_count, start, step);
        break;
      case 2:
        buffer_ptr = init_columnar_buffer<int16_t>(reinterpret_cast<int16_t*>(buffer_ptr),
                                                   init_vals[init_idx++],
                                                   groups_buffer_entry_count,
                                                   start,
                                                   step);
        break;
      case 4:
        buffer_ptr = init_columnar_buffer<int32_t>(reinterpret_cast<int32_t*>(buffer_ptr),
                                                   init_vals[init_idx++],
                                                   groups_buffer_entry_count,
                                                   start,
                                                   step);
        break;
      case 8:
        buffer_ptr = init_columnar_buffer<int64_t>(reinterpret_cast<int64_t*>(buffer_ptr),
                                                   init_vals[init_idx++],
                                                   groups_buffer_entry_count,
                                                   start,
                                                   step);
        break;
      case 0:
        continue;
      default:
        // FIXME(miyu): CUDA linker doesn't accept assertion on GPU yet now.
        break;
    }
  }
  __syncthreads();
}

template <typename K>
inline __device__ void fill_empty_device_key(K* keys_ptr,
                                             const uint32_t key_count,
                                             const K empty_key) {
  for (uint32_t i = 0; i < key_count; ++i) {
    keys_ptr[i] = empty_key;
  }
}

__global__ void init_group_by_buffer_gpu(int64_t* groups_buffer,
                                         const int64_t* init_vals,
                                         const uint32_t groups_buffer_entry_count,
                                         const uint32_t key_count,
                                         const uint32_t key_width,
                                         const uint32_t row_size_quad,
                                         const bool keyless,
                                         const int8_t warp_size) {
  const int32_t start = blockIdx.x * blockDim.x + threadIdx.x;
  const int32_t step = blockDim.x * gridDim.x;
  if (keyless) {
    for (int32_t i = start;
         i < groups_buffer_entry_count * row_size_quad * static_cast<int32_t>(warp_size);
         i += step) {
      groups_buffer[i] = init_vals[i % row_size_quad];
    }
    __syncthreads();
    return;
  }

  for (int32_t i = start; i < groups_buffer_entry_count; i += step) {
    int64_t* keys_ptr = groups_buffer + i * row_size_quad;
    switch (key_width) {
      case 4:
        fill_empty_device_key(
            reinterpret_cast<int32_t*>(keys_ptr), key_count, EMPTY_KEY_32);
        break;
      case 8:
        fill_empty_device_key(
            reinterpret_cast<int64_t*>(keys_ptr), key_count, EMPTY_KEY_64);
        break;
      default:
        break;
    }
  }

  const uint32_t values_off_quad =
      align_to_int64(key_count * key_width) / sizeof(int64_t);
  for (uint32_t i = start; i < groups_buffer_entry_count; i += step) {
    int64_t* vals_ptr = groups_buffer + i * row_size_quad + values_off_quad;
    const uint32_t val_count =
        row_size_quad - values_off_quad;  // value slots are always 64-bit
    for (uint32_t j = 0; j < val_count; ++j) {
      vals_ptr[j] = init_vals[j];
    }
  }
  __syncthreads();
}

__global__ void init_columnar_group_by_buffer_gpu_wrapper(
    int64_t* groups_buffer,
    const int64_t* init_vals,
    const uint32_t groups_buffer_entry_count,
    const uint32_t key_count,
    const uint32_t agg_col_count,
    const int8_t* col_sizes,
    const bool need_padding,
    const bool keyless,
    const int8_t key_size) {
  init_columnar_group_by_buffer_gpu_impl(groups_buffer,
                                         init_vals,
                                         groups_buffer_entry_count,
                                         key_count,
                                         agg_col_count,
                                         col_sizes,
                                         need_padding,
                                         keyless,
                                         key_size);
}

void init_group_by_buffer_on_device(int64_t* groups_buffer,
                                    const int64_t* init_vals,
                                    const uint32_t groups_buffer_entry_count,
                                    const uint32_t key_count,
                                    const uint32_t key_width,
                                    const uint32_t row_size_quad,
                                    const bool keyless,
                                    const int8_t warp_size,
                                    const size_t block_size_x,
                                    const size_t grid_size_x) {
  auto qe_cuda_stream = getQueryEngineCudaStream();
  init_group_by_buffer_gpu<<<grid_size_x, block_size_x, 0, qe_cuda_stream>>>(
      groups_buffer,
      init_vals,
      groups_buffer_entry_count,
      key_count,
      key_width,
      row_size_quad,
      keyless,
      warp_size);
  checkCudaErrors(hipStreamSynchronize(qe_cuda_stream));
}

void init_columnar_group_by_buffer_on_device(int64_t* groups_buffer,
                                             const int64_t* init_vals,
                                             const uint32_t groups_buffer_entry_count,
                                             const uint32_t key_count,
                                             const uint32_t agg_col_count,
                                             const int8_t* col_sizes,
                                             const bool need_padding,
                                             const bool keyless,
                                             const int8_t key_size,
                                             const size_t block_size_x,
                                             const size_t grid_size_x) {
  auto qe_cuda_stream = getQueryEngineCudaStream();
  init_columnar_group_by_buffer_gpu_wrapper<<<grid_size_x,
                                              block_size_x,
                                              0,
                                              qe_cuda_stream>>>(groups_buffer,
                                                                init_vals,
                                                                groups_buffer_entry_count,
                                                                key_count,
                                                                agg_col_count,
                                                                col_sizes,
                                                                need_padding,
                                                                keyless,
                                                                key_size);
  checkCudaErrors(hipStreamSynchronize(qe_cuda_stream));
}
