#include "hip/hip_runtime.h"
/*
 * Copyright 2017 MapD Technologies, Inc.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "HashJoinRuntime.cpp"

#include <thrust/device_ptr.h>
#include <thrust/scan.h>

__global__ void fill_hash_join_buff_wrapper(int32_t* buff,
                                            const int32_t invalid_slot_val,
                                            const JoinColumn join_column,
                                            const JoinColumnTypeInfo type_info,
                                            int* err) {
  int partial_err = SUFFIX(fill_hash_join_buff)(
      buff, invalid_slot_val, join_column, type_info, NULL, NULL, -1, -1);
  atomicCAS(err, 0, partial_err);
}

__global__ void fill_hash_join_buff_bucketized_wrapper(
    int32_t* buff,
    const int32_t invalid_slot_val,
    const JoinColumn join_column,
    const JoinColumnTypeInfo type_info,
    int* err,
    const int64_t bucket_normalization) {
  int partial_err = SUFFIX(fill_hash_join_buff_bucketized)(buff,
                                                           invalid_slot_val,
                                                           join_column,
                                                           type_info,
                                                           NULL,
                                                           NULL,
                                                           -1,
                                                           -1,
                                                           bucket_normalization);
  atomicCAS(err, 0, partial_err);
}

void fill_hash_join_buff_on_device_bucketized(int32_t* buff,
                                              const int32_t invalid_slot_val,
                                              int* dev_err_buff,
                                              const JoinColumn join_column,
                                              const JoinColumnTypeInfo type_info,
                                              const size_t block_size_x,
                                              const size_t grid_size_x,
                                              const int64_t bucket_normalization) {
  fill_hash_join_buff_bucketized_wrapper<<<grid_size_x, block_size_x>>>(
      buff, invalid_slot_val, join_column, type_info, dev_err_buff, bucket_normalization);
}

void fill_hash_join_buff_on_device(int32_t* buff,
                                   const int32_t invalid_slot_val,
                                   int* dev_err_buff,
                                   const JoinColumn join_column,
                                   const JoinColumnTypeInfo type_info,
                                   const size_t block_size_x,
                                   const size_t grid_size_x) {
  fill_hash_join_buff_wrapper<<<grid_size_x, block_size_x>>>(
      buff, invalid_slot_val, join_column, type_info, dev_err_buff);
}

__global__ void fill_hash_join_buff_wrapper_sharded_bucketized(
    int32_t* buff,
    const int32_t invalid_slot_val,
    const JoinColumn join_column,
    const JoinColumnTypeInfo type_info,
    const ShardInfo shard_info,
    int* err,
    const int64_t bucket_normalization) {
  int partial_err = SUFFIX(fill_hash_join_buff_sharded_bucketized)(buff,
                                                                   invalid_slot_val,
                                                                   join_column,
                                                                   type_info,
                                                                   shard_info,
                                                                   NULL,
                                                                   NULL,
                                                                   -1,
                                                                   -1,
                                                                   bucket_normalization);
  atomicCAS(err, 0, partial_err);
}

__global__ void fill_hash_join_buff_wrapper_sharded(int32_t* buff,
                                                    const int32_t invalid_slot_val,
                                                    const JoinColumn join_column,
                                                    const JoinColumnTypeInfo type_info,
                                                    const ShardInfo shard_info,
                                                    int* err) {
  int partial_err = SUFFIX(fill_hash_join_buff_sharded)(
      buff, invalid_slot_val, join_column, type_info, shard_info, NULL, NULL, -1, -1);
  atomicCAS(err, 0, partial_err);
}

void fill_hash_join_buff_on_device_sharded_bucketized(
    int32_t* buff,
    const int32_t invalid_slot_val,
    int* dev_err_buff,
    const JoinColumn join_column,
    const JoinColumnTypeInfo type_info,
    const ShardInfo shard_info,
    const size_t block_size_x,
    const size_t grid_size_x,
    const int64_t bucket_normalization) {
  fill_hash_join_buff_wrapper_sharded_bucketized<<<grid_size_x, block_size_x>>>(
      buff,
      invalid_slot_val,
      join_column,
      type_info,
      shard_info,
      dev_err_buff,
      bucket_normalization);
}

void fill_hash_join_buff_on_device_sharded(int32_t* buff,
                                           const int32_t invalid_slot_val,
                                           int* dev_err_buff,
                                           const JoinColumn join_column,
                                           const JoinColumnTypeInfo type_info,
                                           const ShardInfo shard_info,
                                           const size_t block_size_x,
                                           const size_t grid_size_x) {
  fill_hash_join_buff_wrapper_sharded<<<grid_size_x, block_size_x>>>(
      buff, invalid_slot_val, join_column, type_info, shard_info, dev_err_buff);
}

__global__ void init_hash_join_buff_wrapper(int32_t* buff,
                                            const int32_t hash_entry_count,
                                            const int32_t invalid_slot_val) {
  SUFFIX(init_hash_join_buff)(buff, hash_entry_count, invalid_slot_val, -1, -1);
}

void init_hash_join_buff_on_device(int32_t* buff,
                                   const int32_t hash_entry_count,
                                   const int32_t invalid_slot_val,
                                   const size_t block_size_x,
                                   const size_t grid_size_x) {
  init_hash_join_buff_wrapper<<<grid_size_x, block_size_x>>>(
      buff, hash_entry_count, invalid_slot_val);
}

#define VALID_POS_FLAG 0

__global__ void set_valid_pos_flag(int32_t* pos_buff,
                                   const int32_t* count_buff,
                                   const int32_t entry_count) {
  const int32_t start = threadIdx.x + blockDim.x * blockIdx.x;
  const int32_t step = blockDim.x * gridDim.x;
  for (int32_t i = start; i < entry_count; i += step) {
    if (count_buff[i]) {
      pos_buff[i] = VALID_POS_FLAG;
    }
  }
}

__global__ void set_valid_pos(int32_t* pos_buff,
                              int32_t* count_buff,
                              const int32_t entry_count) {
  const int32_t start = threadIdx.x + blockDim.x * blockIdx.x;
  const int32_t step = blockDim.x * gridDim.x;
  for (int32_t i = start; i < entry_count; i += step) {
    if (VALID_POS_FLAG == pos_buff[i]) {
      pos_buff[i] = !i ? 0 : count_buff[i - 1];
    }
  }
}

template <typename COUNT_MATCHES_FUNCTOR, typename FILL_ROW_IDS_FUNCTOR>
void fill_one_to_many_hash_table_on_device_impl(int32_t* buff,
                                                const int32_t hash_entry_count,
                                                const int32_t invalid_slot_val,
                                                const JoinColumn& join_column,
                                                const JoinColumnTypeInfo& type_info,
                                                const size_t block_size_x,
                                                const size_t grid_size_x,
                                                COUNT_MATCHES_FUNCTOR count_matches_func,
                                                FILL_ROW_IDS_FUNCTOR fill_row_ids_func) {
  int32_t* pos_buff = buff;
  int32_t* count_buff = buff + hash_entry_count;
  hipMemset(count_buff, 0, hash_entry_count * sizeof(int32_t));
  count_matches_func();

  set_valid_pos_flag<<<grid_size_x, block_size_x>>>(
      pos_buff, count_buff, hash_entry_count);

  auto count_buff_dev_ptr = thrust::device_pointer_cast(count_buff);
  thrust::inclusive_scan(
      count_buff_dev_ptr, count_buff_dev_ptr + hash_entry_count, count_buff_dev_ptr);
  set_valid_pos<<<grid_size_x, block_size_x>>>(pos_buff, count_buff, hash_entry_count);
  hipMemset(count_buff, 0, hash_entry_count * sizeof(int32_t));
  fill_row_ids_func();
}

void fill_one_to_many_hash_table_on_device(int32_t* buff,
                                           const HashEntryInfo hash_entry_info,
                                           const int32_t invalid_slot_val,
                                           const JoinColumn& join_column,
                                           const JoinColumnTypeInfo& type_info,
                                           const size_t block_size_x,
                                           const size_t grid_size_x) {
  auto hash_entry_count = hash_entry_info.hash_entry_count;
  auto count_matches_func = [hash_entry_count,
                             grid_size_x,
                             block_size_x,
                             count_buff = buff + hash_entry_count,
                             invalid_slot_val,
                             join_column,
                             type_info] {
    SUFFIX(count_matches)<<<grid_size_x, block_size_x>>>(
        count_buff, invalid_slot_val, join_column, type_info);
  };

  auto fill_row_ids_func = [grid_size_x,
                            block_size_x,
                            buff,
                            hash_entry_count,
                            invalid_slot_val,
                            join_column,
                            type_info] {
    SUFFIX(fill_row_ids)<<<grid_size_x, block_size_x>>>(
        buff, hash_entry_count, invalid_slot_val, join_column, type_info);
  };

  fill_one_to_many_hash_table_on_device_impl(buff,
                                             hash_entry_count,
                                             invalid_slot_val,
                                             join_column,
                                             type_info,
                                             block_size_x,
                                             grid_size_x,
                                             count_matches_func,
                                             fill_row_ids_func);
}

void fill_one_to_many_hash_table_on_device_bucketized(int32_t* buff,
                                                      const HashEntryInfo hash_entry_info,
                                                      const int32_t invalid_slot_val,
                                                      const JoinColumn& join_column,
                                                      const JoinColumnTypeInfo& type_info,
                                                      const size_t block_size_x,
                                                      const size_t grid_size_x) {
  auto hash_entry_count = hash_entry_info.getNormalizedHashEntryCount();
  auto count_matches_func = [grid_size_x,
                             block_size_x,
                             count_buff = buff + hash_entry_count,
                             invalid_slot_val,
                             join_column,
                             type_info,
                             bucket_normalization =
                                 hash_entry_info.bucket_normalization] {
    SUFFIX(count_matches_bucketized)<<<grid_size_x, block_size_x>>>(
        count_buff, invalid_slot_val, join_column, type_info, bucket_normalization);
  };

  auto fill_row_ids_func = [grid_size_x,
                            block_size_x,
                            buff,
                            hash_entry_count =
                                hash_entry_info.getNormalizedHashEntryCount(),
                            invalid_slot_val,
                            join_column,
                            type_info,
                            bucket_normalization = hash_entry_info.bucket_normalization] {
    SUFFIX(fill_row_ids_bucketized)<<<grid_size_x, block_size_x>>>(buff,
                                                                   hash_entry_count,
                                                                   invalid_slot_val,
                                                                   join_column,
                                                                   type_info,
                                                                   bucket_normalization);
  };

  fill_one_to_many_hash_table_on_device_impl(buff,
                                             hash_entry_count,
                                             invalid_slot_val,
                                             join_column,
                                             type_info,
                                             block_size_x,
                                             grid_size_x,
                                             count_matches_func,
                                             fill_row_ids_func);
}

void fill_one_to_many_hash_table_on_device_sharded(int32_t* buff,
                                                   const HashEntryInfo hash_entry_info,
                                                   const int32_t invalid_slot_val,
                                                   const JoinColumn& join_column,
                                                   const JoinColumnTypeInfo& type_info,
                                                   const ShardInfo& shard_info,
                                                   const size_t block_size_x,
                                                   const size_t grid_size_x) {
  auto hash_entry_count = hash_entry_info.hash_entry_count;
  int32_t* pos_buff = buff;
  int32_t* count_buff = buff + hash_entry_count;
  hipMemset(count_buff, 0, hash_entry_count * sizeof(int32_t));
  SUFFIX(count_matches_sharded)<<<grid_size_x, block_size_x>>>(
      count_buff, invalid_slot_val, join_column, type_info, shard_info);

  set_valid_pos_flag<<<grid_size_x, block_size_x>>>(
      pos_buff, count_buff, hash_entry_count);

  auto count_buff_dev_ptr = thrust::device_pointer_cast(count_buff);
  thrust::inclusive_scan(
      count_buff_dev_ptr, count_buff_dev_ptr + hash_entry_count, count_buff_dev_ptr);
  set_valid_pos<<<grid_size_x, block_size_x>>>(pos_buff, count_buff, hash_entry_count);
  hipMemset(count_buff, 0, hash_entry_count * sizeof(int32_t));
  SUFFIX(fill_row_ids_sharded)<<<grid_size_x, block_size_x>>>(
      buff, hash_entry_count, invalid_slot_val, join_column, type_info, shard_info);
}

template <typename T, typename KEY_HANDLER>
void fill_one_to_many_baseline_hash_table_on_device(int32_t* buff,
                                                    const T* composite_key_dict,
                                                    const size_t hash_entry_count,
                                                    const int32_t invalid_slot_val,
                                                    const KEY_HANDLER* key_handler,
                                                    const size_t num_elems,
                                                    const size_t block_size_x,
                                                    const size_t grid_size_x) {
  auto pos_buff = buff;
  auto count_buff = buff + hash_entry_count;
  hipMemset(count_buff, 0, hash_entry_count * sizeof(int32_t));
  count_matches_baseline_gpu<<<grid_size_x, block_size_x>>>(
      count_buff, composite_key_dict, hash_entry_count, key_handler, num_elems);

  set_valid_pos_flag<<<grid_size_x, block_size_x>>>(
      pos_buff, count_buff, hash_entry_count);

  auto count_buff_dev_ptr = thrust::device_pointer_cast(count_buff);
  thrust::inclusive_scan(
      count_buff_dev_ptr, count_buff_dev_ptr + hash_entry_count, count_buff_dev_ptr);
  set_valid_pos<<<grid_size_x, block_size_x>>>(pos_buff, count_buff, hash_entry_count);
  hipMemset(count_buff, 0, hash_entry_count * sizeof(int32_t));
  fill_row_ids_baseline_gpu<<<grid_size_x, block_size_x>>>(buff,
                                                           composite_key_dict,
                                                           hash_entry_count,
                                                           invalid_slot_val,
                                                           key_handler,
                                                           num_elems);
}

template <typename T>
__global__ void init_baseline_hash_join_buff_wrapper(int8_t* hash_join_buff,
                                                     const size_t entry_count,
                                                     const size_t key_component_count,
                                                     const bool with_val_slot,
                                                     const int32_t invalid_slot_val) {
  SUFFIX(init_baseline_hash_join_buff)<T>(hash_join_buff,
                                          entry_count,
                                          key_component_count,
                                          with_val_slot,
                                          invalid_slot_val,
                                          -1,
                                          -1);
}

void init_baseline_hash_join_buff_on_device_32(int8_t* hash_join_buff,
                                               const int32_t entry_count,
                                               const size_t key_component_count,
                                               const bool with_val_slot,
                                               const int32_t invalid_slot_val,
                                               const size_t block_size_x,
                                               const size_t grid_size_x) {
  init_baseline_hash_join_buff_wrapper<int32_t><<<grid_size_x, block_size_x>>>(
      hash_join_buff, entry_count, key_component_count, with_val_slot, invalid_slot_val);
}

void init_baseline_hash_join_buff_on_device_64(int8_t* hash_join_buff,
                                               const int32_t entry_count,
                                               const size_t key_component_count,
                                               const bool with_val_slot,
                                               const int32_t invalid_slot_val,
                                               const size_t block_size_x,
                                               const size_t grid_size_x) {
  init_baseline_hash_join_buff_wrapper<int64_t><<<grid_size_x, block_size_x>>>(
      hash_join_buff, entry_count, key_component_count, with_val_slot, invalid_slot_val);
}

template <typename T, typename KEY_HANDLER>
__global__ void fill_baseline_hash_join_buff_wrapper(int8_t* hash_buff,
                                                     const size_t entry_count,
                                                     const int32_t invalid_slot_val,
                                                     const size_t key_component_count,
                                                     const bool with_val_slot,
                                                     int* err,
                                                     const KEY_HANDLER* key_handler,
                                                     const size_t num_elems) {
  int partial_err = SUFFIX(fill_baseline_hash_join_buff)<T>(hash_buff,
                                                            entry_count,
                                                            invalid_slot_val,
                                                            key_component_count,
                                                            with_val_slot,
                                                            key_handler,
                                                            num_elems,
                                                            -1,
                                                            -1);
  atomicCAS(err, 0, partial_err);
}

void fill_baseline_hash_join_buff_on_device_32(int8_t* hash_buff,
                                               const size_t entry_count,
                                               const int32_t invalid_slot_val,
                                               const size_t key_component_count,
                                               const bool with_val_slot,
                                               int* dev_err_buff,
                                               const GenericKeyHandler* key_handler,
                                               const size_t num_elems,
                                               const size_t block_size_x,
                                               const size_t grid_size_x) {
  fill_baseline_hash_join_buff_wrapper<int32_t>
      <<<grid_size_x, block_size_x>>>(hash_buff,
                                      entry_count,
                                      invalid_slot_val,
                                      key_component_count,
                                      with_val_slot,
                                      dev_err_buff,
                                      key_handler,
                                      num_elems);
}

void fill_baseline_hash_join_buff_on_device_64(int8_t* hash_buff,
                                               const size_t entry_count,
                                               const int32_t invalid_slot_val,
                                               const size_t key_component_count,
                                               const bool with_val_slot,
                                               int* dev_err_buff,
                                               const GenericKeyHandler* key_handler,
                                               const size_t num_elems,
                                               const size_t block_size_x,
                                               const size_t grid_size_x) {
  fill_baseline_hash_join_buff_wrapper<unsigned long long>
      <<<grid_size_x, block_size_x>>>(hash_buff,
                                      entry_count,
                                      invalid_slot_val,
                                      key_component_count,
                                      with_val_slot,
                                      dev_err_buff,
                                      key_handler,
                                      num_elems);
}

void overlaps_fill_baseline_hash_join_buff_on_device_64(
    int8_t* hash_buff,
    const size_t entry_count,
    const int32_t invalid_slot_val,
    const size_t key_component_count,
    const bool with_val_slot,
    int* dev_err_buff,
    const OverlapsKeyHandler* key_handler,
    const size_t num_elems,
    const size_t block_size_x,
    const size_t grid_size_x) {
  fill_baseline_hash_join_buff_wrapper<unsigned long long>
      <<<grid_size_x, block_size_x>>>(hash_buff,
                                      entry_count,
                                      invalid_slot_val,
                                      key_component_count,
                                      with_val_slot,
                                      dev_err_buff,
                                      key_handler,
                                      num_elems);
}

void fill_one_to_many_baseline_hash_table_on_device_32(
    int32_t* buff,
    const int32_t* composite_key_dict,
    const size_t hash_entry_count,
    const int32_t invalid_slot_val,
    const size_t key_component_count,
    const GenericKeyHandler* key_handler,
    const size_t num_elems,
    const size_t block_size_x,
    const size_t grid_size_x) {
  fill_one_to_many_baseline_hash_table_on_device<int32_t>(buff,
                                                          composite_key_dict,
                                                          hash_entry_count,
                                                          invalid_slot_val,
                                                          key_handler,
                                                          num_elems,
                                                          block_size_x,
                                                          grid_size_x);
}

void fill_one_to_many_baseline_hash_table_on_device_64(
    int32_t* buff,
    const int64_t* composite_key_dict,
    const size_t hash_entry_count,
    const int32_t invalid_slot_val,
    const GenericKeyHandler* key_handler,
    const size_t num_elems,
    const size_t block_size_x,
    const size_t grid_size_x) {
  fill_one_to_many_baseline_hash_table_on_device<int64_t>(buff,
                                                          composite_key_dict,
                                                          hash_entry_count,
                                                          invalid_slot_val,
                                                          key_handler,
                                                          num_elems,
                                                          block_size_x,
                                                          grid_size_x);
}

void overlaps_fill_one_to_many_baseline_hash_table_on_device_64(
    int32_t* buff,
    const int64_t* composite_key_dict,
    const size_t hash_entry_count,
    const int32_t invalid_slot_val,
    const OverlapsKeyHandler* key_handler,
    const size_t num_elems,
    const size_t block_size_x,
    const size_t grid_size_x) {
  fill_one_to_many_baseline_hash_table_on_device<int64_t>(buff,
                                                          composite_key_dict,
                                                          hash_entry_count,
                                                          invalid_slot_val,
                                                          key_handler,
                                                          num_elems,
                                                          block_size_x,
                                                          grid_size_x);
}

void approximate_distinct_tuples_on_device_overlaps(uint8_t* hll_buffer,
                                                    const uint32_t b,
                                                    int32_t* row_counts_buffer,
                                                    const OverlapsKeyHandler* key_handler,
                                                    const size_t num_elems,
                                                    const size_t block_size_x,
                                                    const size_t grid_size_x) {
  approximate_distinct_tuples_impl_gpu<<<grid_size_x, block_size_x>>>(
      hll_buffer, row_counts_buffer, b, num_elems, key_handler);

  auto row_counts_buffer_ptr = thrust::device_pointer_cast(row_counts_buffer);
  thrust::inclusive_scan(
      row_counts_buffer_ptr, row_counts_buffer_ptr + num_elems, row_counts_buffer_ptr);
}

void approximate_distinct_tuples_on_device(uint8_t* hll_buffer,
                                           const uint32_t b,
                                           const GenericKeyHandler* key_handler,
                                           const size_t num_elems,
                                           const size_t block_size_x,
                                           const size_t grid_size_x) {
  approximate_distinct_tuples_impl_gpu<<<grid_size_x, block_size_x>>>(
      hll_buffer, nullptr, b, num_elems, key_handler);
}

void compute_bucket_sizes_on_device(double* bucket_sizes_buffer,
                                    const JoinColumn* join_column,
                                    const double bucket_sz_threshold,
                                    const size_t block_size_x,
                                    const size_t grid_size_x) {
  compute_bucket_sizes_impl_gpu<2><<<grid_size_x, block_size_x>>>(
      bucket_sizes_buffer, join_column, bucket_sz_threshold, block_size_x, grid_size_x);
}
